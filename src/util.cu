#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "util.h"
#include "error.h"

ullong vseed;
ullong vran;
hiprandGenerator_t gen;
int *_derrno;
int _herrno;

//=================================================
// random number generator
//=================================================
void ran_seed(long j){
  vseed = j;  vran = 4101842887655102017LL;
  vran ^= vseed;
  vran ^= vran >> 21; vran ^= vran << 35; vran ^= vran >> 4;
  vran = vran * 2685821657736338717LL;
}

float ran_ran2(){
    vran ^= vran >> 21; vran ^= vran << 35; vran ^= vran >> 4;
    ullong t = vran * 2685821657736338717LL; return 5.42101086242752217e-20*t;
}

void initializeDevice(int device){
    hipSetDevice(device);
    CSAFECALL( hipMalloc((void**)&_derrno, sizeof(int)) );
    CSAFECALL( hipMemset(_derrno, 0, sizeof(int)) );
}

void destroyDevice(){
    hiprandDestroyGenerator(gen);
    hipDeviceReset();
}

void setSeed(int seed){
    ran_seed(seed);
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    //hiprandDestroyGenerator(gen);
}

float *createDeviceArray(int len){
    float *out;
    CSAFECALL( hipMalloc((void**) &out, sizeof(float)*len) );
    hipMemset(out, 0, sizeof(float)*len);
    return out;
}
double *createDeviceArrayD(int len){
    double *out;
    CSAFECALL( hipMalloc((void**) &out, sizeof(double)*len) );
    hipMemset(out, 0, sizeof(double)*len);
    return out;
}
cfloat *createDeviceArrayC(int len){
    cfloat *out;
    CSAFECALL( hipMalloc((void**) &out, sizeof(cfloat)*len) );
    hipMemset(out, 0, sizeof(cfloat)*len);
    return out;
}
int *createDeviceArrayInt(int len){
    int *out;
    CSAFECALL( hipMalloc((void**) &out, sizeof(int)*len) );
    hipMemset(out, 0, sizeof(int)*len);
    return out;
}

void freeDeviceArray(float *f){ CSAFECALL(hipFree(f)); }
void freeDeviceArrayD(double *f){ CSAFECALL(hipFree(f)); }
void freeDeviceArrayC(cfloat *f){ CSAFECALL(hipFree(f)); }
void freeDeviceArrayInt(int *f){ CSAFECALL(hipFree(f)); }

void copyToDevice(int total_len, float *inpy, float *dev){
    CSAFECALL( hipMemcpy(dev, inpy, sizeof(float)*total_len, hipMemcpyHostToDevice) );
}

void copyFromDevice(float *dev, int total_len, float *onpy){
    CSAFECALL( hipMemcpy(onpy, dev, sizeof(float)*total_len, hipMemcpyDeviceToHost) );
}

void copyToDeviceD(int total_len, double *inpy, double *dev){
    CSAFECALL( hipMemcpy(dev, inpy, sizeof(double)*total_len, hipMemcpyHostToDevice) );
}

void copyFromDeviceD(double *dev, int total_len, double *onpy){
    CSAFECALL( hipMemcpy(onpy, dev, sizeof(double)*total_len, hipMemcpyDeviceToHost) );
}

void copyToDeviceInt(int total_len, int *inpy, int *dev){
    CSAFECALL( hipMemcpy(dev, inpy, sizeof(int)*total_len, hipMemcpyHostToDevice) );
}

void copyFromDeviceInt(int *dev, int total_len, int *onpy){
    CSAFECALL( hipMemcpy(onpy, dev, sizeof(int)*total_len, hipMemcpyDeviceToHost) );
}

void copyToDeviceC(int total_len, cfloat *inpy, cfloat *dev){
    CSAFECALL( hipMemcpy(dev, inpy, sizeof(cfloat)*total_len, hipMemcpyHostToDevice) );
}

void copyFromDeviceC(cfloat *dev, int total_len, cfloat *onpy){
    CSAFECALL( hipMemcpy(onpy, dev, sizeof(cfloat)*total_len, hipMemcpyDeviceToHost) );
}


void copyArray(float *src, int len, float *dst){
    memcpy(dst, src, sizeof(float)*len);
}

void copyArrayInt(int *src, int len, int *dst){
    memcpy(dst, src, sizeof(int)*len);
}
